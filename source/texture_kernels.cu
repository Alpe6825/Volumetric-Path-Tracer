#include "hip/hip_runtime.h"
//--------------------------------------------------------------------------------
//
//	Redistribution and use in source and binary forms, with or without
//	modification, are permitted provided that the following conditions are met :
//
//	*Redistributions of source code must retain the above copyright notice, this
//	list of conditions and the following disclaimer.
//
//	* Redistributions in binary form must reproduce the above copyright notice,
//	this list of conditions and the following disclaimer in the documentation
//	and/or other materials provided with the distribution.
//	
//	* Neither the name of the copyright holder nor the names of its
//	contributors may be used to endorse or promote products derived from
//	this software without specific prior written permission.
//	
//	THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
//	AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
//	IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
//	DISCLAIMED.IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
//	FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
//	DAMAGES(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
//	SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
//	CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
//	OR TORT(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
//	OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
// 
// Copyright(c) 2019, Sergen Eren
// All rights reserved.
//----------------------------------------------------------------------------------
// 
//	Version 1.0: Sergen Eren, 02/11/2019
//
// File: Kernels to calculate and load the procedural sky value and cdf textures
//
//-----------------------------------------------

#define _USE_MATH_DEFINES
#include <cmath>
#include <stdio.h>
#include <float.h>

// Cuda includes
#include <hip/hip_runtime.h> 
#include <hiprand/hiprand_kernel.h>
#include <>
#include "hip/hip_vector_types.h"


// Internal includes
#include "kernel_params.h"
#include "cuda_noise.cuh"

#define INV_2_PI		1.0f / (2.0f * M_PI) 
#define INV_4_PI		1.0f / (4.0f * M_PI) 
#define INV_PI			1.0f / M_PI 


typedef hiprandStatePhilox4_32_10_t Rand_state;
#define rand(state) hiprand_uniform(state)



extern "C" __global__ void glow(const Kernel_params kernel_params, float treshold , const int width, const int height) {

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= width || y >= height) return;
	const unsigned int idx = y * width + x;

	// TODO gaussian blur and add glow effect to display buffer 


}

extern "C" __global__ void fill_volume_buffer( float *buffer, const int3 dims, const float scale, const int noise_type) {

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int z = blockIdx.z * blockDim.z + threadIdx.z;

	if (x >= dims.x || y >= dims.y || z >= dims.z) return;

	const unsigned int idx = x + dims.x * (y + dims.y * z);

	Rand_state rand_state;

	int seed = 123;
	float du = 1.0f / (float)dims.x;

	float dx = cudaNoise::randomFloat(482 + floor(rand(&rand_state) * 2) * 47 + seed) / (float)dims.x;
	float dy = cudaNoise::randomFloat(472 + floor(rand(&rand_state) * 2) * 38 + seed) / (float)dims.y;
	float dz = cudaNoise::randomFloat(348 + floor(rand(&rand_state) * 2) * 14 + seed) / (float)dims.z;

	float3 pos = make_float3(x+dx, y+dy, z+dz);
	
	switch (noise_type)
	{
	case(0):
		buffer[idx] = cudaNoise::perlinNoise(pos, scale, seed);
		break;
	case(1):
		buffer[idx] = cudaNoise::simplexNoise(pos, scale, seed);
		break;
	case(2):
		buffer[idx] = cudaNoise::worleyNoise(pos, scale, seed, 300.1f, 4, 4, 1.0f);
		break;
	case(3):
		buffer[idx] = cudaNoise::repeaterPerlin(pos, scale, seed, 128, 1.9f, 0.5f);
		break;
	case(4):
		buffer[idx] = cudaNoise::repeaterPerlinAbs(pos, scale, seed, 128, 1.9f, 0.5f);
		break;
	case(5):
		buffer[idx] = cudaNoise::fractalSimplex(pos, scale, seed, du, 512, 1.5f, 0.95f);
		break;
	case(6):
		buffer[idx] = cudaNoise::repeaterTurbulence(pos, 0.2f, scale, seed, 0.8f, 32, cudaNoise::BASIS_PERLIN, cudaNoise::BASIS_PERLIN);
		break;
	case(7):
		buffer[idx] = cudaNoise::cubicValue(pos, scale, seed);
		break;
	case(8):
		buffer[idx] = cudaNoise::spots(pos, scale, seed, 0.1f, 0, 8, 1.0f, cudaNoise::SHAPE_STEP);
		break;
	}

}
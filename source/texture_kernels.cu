#include "hip/hip_runtime.h"
//--------------------------------------------------------------------------------
//
//	Redistribution and use in source and binary forms, with or without
//	modification, are permitted provided that the following conditions are met :
//
//	*Redistributions of source code must retain the above copyright notice, this
//	list of conditions and the following disclaimer.
//
//	* Redistributions in binary form must reproduce the above copyright notice,
//	this list of conditions and the following disclaimer in the documentation
//	and/or other materials provided with the distribution.
//	
//	* Neither the name of the copyright holder nor the names of its
//	contributors may be used to endorse or promote products derived from
//	this software without specific prior written permission.
//	
//	THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
//	AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
//	IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
//	DISCLAIMED.IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
//	FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
//	DAMAGES(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
//	SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
//	CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
//	OR TORT(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
//	OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
// 
// Copyright(c) 2019, Sergen Eren
// All rights reserved.
//----------------------------------------------------------------------------------
// 
//	Version 1.0: Sergen Eren, 02/11/2019
//
// File: Kernels to calculate and load the procedural sky value and cdf textures
//
//-----------------------------------------------

#define _USE_MATH_DEFINES
#include <cmath>
#include <stdio.h>
#include <float.h>

// Cuda includes
#include <hip/hip_runtime.h> 
#include <hiprand/hiprand_kernel.h>
#include <>
#include "hip/hip_vector_types.h"


// Internal includes
#include "kernel_params.h"

#define INV_2_PI		1.0f / (2.0f * M_PI) 
#define INV_4_PI		1.0f / (4.0f * M_PI) 
#define INV_PI			1.0f / M_PI 


__device__ inline bool solveQuadratic(
	float a,
	float b,
	float c,
	float& x1,
	float& x2)
{
	if (b == 0) {
		// Handle special case where the the two vector ray.dir and V are perpendicular
		// with V = ray.orig - sphere.centre
		if (a == 0) return false;
		x1 = 0; x2 = sqrt(-c / a);
		return true;
	}

	float discr = b * b - 4 * a * c;

	if (discr < 0) return false;

	float q = (b < 0.f) ? -0.5f * (b - sqrt(discr)) : -0.5f * (b + sqrt(discr));
	x1 = q / a;
	x2 = c / q;

	return true;
}


__device__ bool raySphereIntersect(
	const float3& orig,
	const float3& dir,
	const float& radius,
	float& t0,
	float& t1)
{

	float A = squared_length(dir);
	float B = 2 * (dir.x * orig.x + dir.y * orig.y + dir.z * orig.z);
	float C = orig.x * orig.x + orig.y * orig.y + orig.z * orig.z - radius * radius;

	if (!solveQuadratic(A, B, C, t0, t1)) return false;

	if (t0 > t1) {
		float tempt = t1;
		t1 = t0;
		t0 = tempt;
	}
	return true;
}


__device__ inline float degree_to_radians(
	float degree)
{

	return degree * M_PI / 180.0f;

}


__device__ inline float3 degree_to_cartesian(
	float azimuth,
	float elevation)
{

	float az = clamp(azimuth, .0f, 360.0f);
	float el = clamp(elevation, .0f, 90.0f);

	az = degree_to_radians(az);
	el = degree_to_radians(90.0f - el);

	float x = sinf(el) * cosf(az);
	float y = cosf(el);
	float z = sinf(el) * sinf(az);

	return normalize(make_float3(x, y, z));
}

__device__ inline float3 sample_atmosphere(
	const Kernel_params &kernel_params,
	const float3 orig,
	const float3 dir,
	const float3 intensity)
{

	// initial parameters
	float	atmosphereRadius = 6420e3f;
	float3	sunDirection = degree_to_cartesian(kernel_params.azimuth, kernel_params.elevation);
	float	earthRadius = 6360e3f;
	float	Hr = 7994.0f;
	float	Hm = 1200.0f;
	float3	betaR = make_float3(3.8e-6f, 13.5e-6f, 33.1e-6f);
	float3	betaM = make_float3(21e-6f);
	//


	float t0, t1;
	float tmin, tmax = FLT_MAX;
	float3 pos = orig;
	pos.y += 1000 + 6360e3f;

	if (raySphereIntersect(pos, dir, 6360e3f, t0, t1) && t1 > .0f) tmax = fmaxf(.0f, t0);
	tmin = .0f;
	if (!raySphereIntersect(pos, dir, atmosphereRadius, t0, t1) || t1 < 0) return make_float3(1.0f, .0f, .0f);
	if (t0 > tmin && t0 > 0) tmin = t0;
	if (t1 < tmax) tmax = t1;

	uint numSamples = 16;
	uint numSamplesLight = 8;

	float segmentLength = (tmax - tmin) / numSamples;
	float tCurrent = tmin;
	float3 sumR = make_float3(0.0f, .0f, .0f); // Rayleigh contribution
	float3 sumM = make_float3(0.0f, .0f, .0f); // Mie contribution

	float opticalDepthR = 0, opticalDepthM = 0;
	float mu = dot(dir, sunDirection); // mu in the paper which is the cosine of the angle between the sun direction and the ray direction
	float phaseR = 3.f / (16.f * M_PI) * (1 + mu * mu);
	float g = 0.76f;

	float phaseM = 3.f / (8.f * M_PI) * ((1.f - g * g) * (1.f + mu * mu)) / ((2.f + g * g) * pow(1.f + g * g - 2.f * g * mu, 1.5f));

	for (uint i = 0; i < numSamples; ++i) {
		float3 samplePosition = pos + (tCurrent + segmentLength * 0.5f) * dir;
		float height = length(samplePosition) - earthRadius;
		// compute optical depth for light
		float hr = exp(-height / Hr) * segmentLength;
		float hm = exp(-height / Hm) * segmentLength;
		opticalDepthR += hr;
		opticalDepthM += hm;
		// light optical depth
		float t0Light, t1Light;
		raySphereIntersect(samplePosition, sunDirection, atmosphereRadius, t0Light, t1Light);
		float segmentLengthLight = t1Light / numSamplesLight, tCurrentLight = 0;
		float opticalDepthLightR = 0, opticalDepthLightM = 0;
		uint j;
		for (j = 0; j < numSamplesLight; ++j) {
			float3 samplePositionLight = samplePosition + (tCurrentLight + segmentLengthLight * 0.5f) * sunDirection;
			float heightLight = length(samplePositionLight) - earthRadius;
			if (heightLight < 0) break;
			opticalDepthLightR += exp(-heightLight / Hr) * segmentLengthLight;
			opticalDepthLightM += exp(-heightLight / Hm) * segmentLengthLight;
			tCurrentLight += segmentLengthLight;
		}
		if (j == numSamplesLight) {
			float3 tau = betaR * (opticalDepthR + opticalDepthLightR) + betaM * 1.1f * (opticalDepthM + opticalDepthLightM);
			float3 attenuation = make_float3(exp(-tau.x), exp(-tau.y), exp(-tau.z));
			sumR += attenuation * hr;
			sumM += attenuation * hm;
		}
		tCurrent += segmentLength;
	}


	return (sumR * betaR * phaseR + sumM * betaM * phaseM) * intensity;
}




extern "C" __global__ void calculate_textures(const Kernel_params kernel_params, const int width, const int height) {


	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= width || y >= height) return;
	const unsigned int idx = y * kernel_params.resolution.x + x;
	





}
#include "hip/hip_runtime.h"
//--------------------------------------------------------------------------------
//
//	Redistribution and use in source and binary forms, with or without
//	modification, are permitted provided that the following conditions are met :
//
//	*Redistributions of source code must retain the above copyright notice, this
//	list of conditions and the following disclaimer.
//
//	* Redistributions in binary form must reproduce the above copyright notice,
//	this list of conditions and the following disclaimer in the documentation
//	and/or other materials provided with the distribution.
//	
//	* Neither the name of the copyright holder nor the names of its
//	contributors may be used to endorse or promote products derived from
//	this software without specific prior written permission.
//	
//	THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
//	AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
//	IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
//	DISCLAIMED.IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
//	FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
//	DAMAGES(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
//	SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
//	CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
//	OR TORT(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
//	OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
// 
// Copyright(c) 2019, Sergen Eren
// All rights reserved.
//----------------------------------------------------------------------------------
// 
//	Version 1.0.1: Sergen Eren, 18/12/2019
//
// File: Geometry creation and processing kernels
//
//-----------------------------------------------

#include <>
#include <hip/hip_runtime.h> 
#include <hiprand/hiprand_kernel.h>

#include <stdio.h>

#include "geometry/geometry.h"

extern "C" __global__ void create_geometry_list(geometry **d_list, geometry **d_geo_list){

	if (threadIdx.x == 0 && blockIdx.x == 0) {
		
		float3 center = make_float3(100, 320, -200);
		float radius = 100;

		d_list[0] = new sphere(center , radius, make_float3(0.18f), .001f);
		*d_geo_list = new geometry_list(d_list, 1);

	}
}